#include "hip/hip_runtime.h"
// This program implements 2D convolution using Constant memory in CUDA
// By: Nick from CoffeeBeforeArch

#include <cassert>
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <string>
#include <algorithm>


// 3 convolutional mask
#define MASK_DIM 3

// Amount the the matrix will hang over the matrix
#define MASK_OFFSET (MASK_DIM / 2)

// Allocate mask in constant memory
struct mask{
    int x1 [MASK_DIM] = {1,2,1};
    int x2 [MASK_DIM] = {-1,0,1};
    int y1 [MASK_DIM] = {1,0,-1};
    int y2 [MASK_DIM] = {1,2,1};
};


__constant__ struct mask gpuMask;
// 2D Convolution Kernel
// Takes:
//  matrix: Input matrix
//  result: Convolution result
//  N:      Dimensions of the matrices
__global__ void convolution_2d(int *matrix, int *resultX, int *resultY,
                                 float *resultFinal, int N) {
    // Calculate the global thread positions
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Starting index for calculation
    int start_r = row - MASK_OFFSET;
    int start_c = col - MASK_OFFSET;

    // Temp value for accumulating the result
    int temp = 0;

    // Sum(X1b,𝐴 [i,𝑗−𝑏])
    for (int i = 0; i < MASK_DIM; i++) {
        // Range check for cols
        if ((start_c + i) >= 0 && (start_c + i) < N) {
            // Accumulate result
            temp += matrix[(start_r) * N + (start_c + i)] *
                    gpuMask.x1[i];
        }
    }
    // Write back the result
    resultX[row * N + col] = temp;

    //wait for all the threads to write their result
    __syncthreads(); 

    //Sum( X2𝑎, 𝐻 [𝑖−𝑎,𝑗])
    //NOT COALESCED READING!!! this is baaad
    for (int i = 0; i < MASK_DIM; i++) {
        // Range check for cols
        if ((start_r + i) >= 0 && (start_r + i) < N) {
            // Accumulate result
            temp += resultX[(start_r+i) * N + (start_c)] *
                    gpuMask.x2[i];
        }
    }

    // Write back the result
    resultX[row * N + col] = temp;
    //wait for all the threads to write their result
    __syncthreads(); 

    // Sum(Y1b,𝐴 [i,𝑗−𝑏])
    for (int i = 0; i < MASK_DIM; i++) {
        // Range check for cols
        if ((start_c + i) >= 0 && (start_c + i) < N) {
            // Accumulate result
            temp += matrix[(start_r) * N + (start_c + i)] *
                    gpuMask.y1[i];
        }
    }

    resultY[row*N + col] = temp;
    //wait for all the threads to write their result
    __syncthreads(); 

    //Sum( X2𝑎, 𝐻 [𝑖−𝑎,𝑗])
    //NOT COALESCED READING!!! this is baaad
    for (int i = 0; i < MASK_DIM; i++) {
        // Range check for cols
        if ((start_r + i) >= 0 && (start_r + i) < N) {
            // Accumulate result
            temp += resultY[(start_r+i) * N + (start_c)] *
                    gpuMask.y2[i];
        }
    }

    resultY[row*N + col] = temp;
    //wait for all the threads to write their result
    __syncthreads(); 

    //√(𝐻 𝑖𝑗)² + (𝑉 𝑖𝑗)²

    resultFinal[row*N+col] =  sqrt( pow(resultX[row*N+col],2) + pow(resultY[row*N+col],2)) ;
}
    




// Verifies the 2D convolution result on the CPU
// Takes:
//  m:      Original matrix
//  mask:   Convolutional mask
//  result: Result from the GPU
//  N:      Dimensions of the matrix
void verify_result(int *m, int *mask, int *result, int N) {
    // Temp value for accumulating results
    int temp;

    // Intermediate value for more readable code
    int offset_r;
    int offset_c;

    // Go over each row
    for (int i = 0; i < N; i++) {
        // Go over each column
        for (int j = 0; j < N; j++) {
            // Reset the temp variable
            temp = 0;

            // Go over each mask row
            for (int k = 0; k < MASK_DIM; k++) {
            // Update offset value for row
                offset_r = i - MASK_OFFSET + k;

                // Go over each mask column
                for (int l = 0; l < MASK_DIM; l++) {
                    // Update offset value for column
                    offset_c = j - MASK_OFFSET + l;

                    // Range checks if we are hanging off the matrix
                    if (offset_r >= 0 && offset_r < N) {
                        if (offset_c >= 0 && offset_c < N) {
                            // Accumulate partial results
                            temp += m[offset_r * N + offset_c] * mask[k * MASK_DIM + l];
                        }
                    }
                }
            }
            // Fail if the results don't match
            assert(result[i * N + j] == temp);
        }
    }
}




int main() {

    //read image name from user
    char  image_name[512];
    printf("Name the image, with the extension:\n");
    scanf(image_name, 512, stdin);
    sprintf(image_name,"home/guilemon/CudaProjects/Images/%s",image_name);
    printf("%s\n",image_name);

    //load image into cpu memory
    cv :: Mat image = cv :: imread(image_name,cv::IMREAD_GRAYSCALE);
    //error check
    if (image.empty()) {
        std::cout << "Error: Unable to read the image." << std::endl;
        return -1;
    }
        
    // Dimensions of the image
    int N = image.cols * image.rows;

    // Size of the matrix (in bytes)
    size_t bytes_n = N  * sizeof(int);

    // Allocate the matrix and initialize it
    int *matrix = new int[N];
    int *resultX = new int[N];
    int *resultY = new int[N];
    int *resultFinal = new int[N];

    // Allocate device memory
    int *d_matrix;
    int *d_resultX;
    int *d_resultY;
    float *d_resultFinal;

    hipMalloc(&d_matrix, bytes_n);
    hipMalloc(&d_resultX, bytes_n);
    hipMalloc(&d_resultY, bytes_n);
    hipMalloc(&d_resultFinal, bytes_n);
    //allocate memory in gpu for mask
    mask hostMask;
    hipMemcpyToSymbol(HIP_SYMBOL(gpuMask),&hostMask,sizeof(mask));

    // Copy data to the device
    hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);

    // Calculate grid dimensions
    int THREADS = 16;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    // Dimension launch arguments
    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(BLOCKS, BLOCKS);

    // Perform 2D Convolution
    convolution_2d<<<grid_dim, block_dim>>>(d_matrix, d_resultX, d_resultY, d_resultFinal, N);

    // Copy the result back to the CPU
    hipMemcpy(resultFinal, d_resultFinal, bytes_n, hipMemcpyDeviceToHost);


    std::cout << "COMPLETED SUCCESSFULLY!";

    // Free the memory we allocated
    delete[] matrix;
    delete[] resultX;
    delete[] resultY;
    delete[] resultFinal;

    hipFree(d_matrix);
    hipFree(d_resultX);
    hipFree(d_resultY);
    hipFree(d_resultFinal);

    return 0;
}