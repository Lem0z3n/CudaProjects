#include "hip/hip_runtime.h"
// This program implements 2D convolution using Constant memory in CUDA
// By: Nick from CoffeeBeforeArch

#include <cassert>
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <string>
#include <algorithm>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

// 3 convolutional mask
#define MASK_DIM 3

// Amount the the matrix will hang over the matrix
#define MASK_OFFSET (MASK_DIM / 2)



// 2D Convolution Kernel
// Takes:
//  matrix: Input matrix
//  result: Convolution result
//  N:      Dimensions of the matrices
__global__ void sobelOperator(int *matrix, int *gpuMask[], int *resultX, int *resultY,
                                 int *resultFinal, int cols, int rows) { 
                                    //missing rows and cols i cant use N
   
    // Calculate the global thread positions
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    int tCol = tid % cols;
    int tRow = tid / cols;

    // Starting index for calculation
    int start_c = tCol - MASK_OFFSET;

    // Temp value for accumulating the result
    int tempX = 0;
    int tempY = 0;


    // Sum(X1b,𝐴 [i,𝑗−𝑏])
    for (int i = 0; i < MASK_DIM; i++) {
        // Range check for cols
        if ((start_c + i) >= 0 && (start_c + i) < cols) {
            // Accumulate result
            tempX += matrix[(tRow) * cols + (start_c + i)] *
                    gpuMask[2][i];
            tempY +=matrix[(tRow) * cols + (start_c + i)] *
                    gpuMask[4][i];
        }
    }
    // Write back the result
    resultX[tRow * cols + tCol] = tempX;
    resultY[tRow * cols + tCol] = tempY;
    //wait for all the threads to write their result
    __syncthreads(); 

    for (int i = 0; i < MASK_DIM; i++) {
        // Range check for cols
        if ((start_c + i) >= 0 && (start_c + i) < cols) {
            // Accumulate result
            tempX += resultX[(tRow) * cols + (start_c + i)] *
                    gpuMask[1][i];
            tempY +=resultY[(tRow) * cols + (start_c + i)] *
                    gpuMask[3][i];
        }
    }
    // Write back the result
    resultX[tRow * cols + tCol] = tempX;
    resultY[tRow * cols + tCol] = tempY;
    __syncthreads(); 
    
    //√(𝐻 𝑖𝑗)² + (𝑉 𝑖𝑗)²
    float accResult =  sqrt( pow(resultX[tRow*cols+tCol],2) + pow(resultY[tRow*cols+tCol],2));
   
   //if the result is bigger than the threshold write white if not black.
   (accResult>0) ? resultFinal[tRow*cols+tCol] = 255 : resultFinal[tRow*cols+tCol] =0;

    }

bool check_result(int * endRes, char* filename, int columns, int N){

    char name[512];
    sprintf(name,"%s.txt",filename);
    FILE * file = fopen(name,"w");

    int i = 0;
    printf("writing image\n");
    while(fprintf(file," %i ",endRes[i])>0 && i < N){
        i++;
        if(i%columns == 0)
            fprintf(file,"\n");
    }
    fclose(file);

    return true;
}
    
int main(int argc, char * args[]) {

    if(argc < 2){
        printf("please provide image filename\n");
        exit(1);
    }
    //load image into cpu memory
    cv :: Mat image = cv :: imread(args[1],cv::IMREAD_GRAYSCALE);
    //error check
    if (image.empty()) {
        std::cout << "Error: Unable to read the image." << std::endl;
        return -1;
    }
    printf("image read.\n");   
    // Dimensions of the image
    int N = image.cols * image.rows;

    // Size of the matrix (in bytes)
    size_t bytes_n = N  * sizeof(int);
    size_t bytes_res = N * sizeof(int);

    // Allocate the matrix and initialize it
    int *matrix = new int[N];
    int *resultX = new int[N];
    int *resultY = new int[N];
    int *resultFinal = new int[N];
    
    const int mask[4][3] = {{1,2,1},
                            {-1,0,1},
                            {1,0,-1},
                            {1,2,1}};

    //convertin from cv datatype to int[]
    for(int i = 0; i < image.rows; i++){
        for(int j = 0; j < image.cols; j++){
            matrix[i*image.rows+j] = static_cast<int>(image.at<uchar>(i,j));
        }
    }
    

    // Allocate device memory
    int *d_matrix;
    int *d_resultX;
    int *d_resultY;
    int **d_mask;
    int *d_resultFinal;

    hipMalloc(&d_matrix, bytes_n);
    hipMalloc(&d_resultX, bytes_n);
    hipMalloc(&d_resultY, bytes_n);
    hipMalloc(&d_resultFinal, bytes_res);
    hipMalloc(&d_mask, sizeof(mask));

    
    printf("checking image\n");
    check_result(matrix,"matrix",image.cols, N);
    // Copy data to the device
    hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, sizeof(mask), hipMemcpyHostToDevice);
    printf("Image copied to GPU\n");
    
    // Threads per TB
    int THREADS = 256;

    // Number of TBs
    int GRID = (N + THREADS - 1) / THREADS;

    // Perform 2D Convolution

    printf("calling gpu\n");
    sobelOperator<<<GRID, THREADS>>>(d_matrix, d_mask, d_resultX, d_resultY, d_resultFinal, image.cols, image.rows);
    printf("returning from gpu\n");
    // Copy the result back to the CPU

    hipMemcpy(resultFinal, d_resultFinal, bytes_res, hipMemcpyDeviceToHost);

    printf("COMPLETED SUCCESSFULLY!\n");

    check_result(resultFinal,"result",image.cols, N);

    cv :: Mat imageResult(image.cols, image.rows, CV_32F, resultFinal);

    imageResult.convertTo(imageResult, CV_8U);

    char resultName [1024];
    sprintf(resultName, "Completed%s", args[1]);

    if (cv::imwrite(resultName, imageResult))
        std::cout << "Image saved successfully!" << std::endl;
    else
        std::cerr << "Error saving image" << std::endl;
    


    // Free the memory we allocated

    //delete[] matrix;
    delete[] resultX;
    delete[] resultY;
    delete[] resultFinal;
    delete[] matrix;


    hipFree(d_matrix);
    hipFree(d_resultX);
    hipFree(d_resultY);
    hipFree(d_resultFinal);

    return 0;
}