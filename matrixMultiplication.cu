// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <time.h> 

using std::cout;
using std::generate;
using std::vector;

__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

// Check result on the CPU
bool check_result(vector<int> endRes, int N){


    FILE * file = fopen("doesItWrite.txt","w");

    char buf[sizeof(int)+4];
    int i = 0;
    printf("writing image\n");
    while(fprintf(file," %f ",endRes[i])>0 && i < N){
        i++;
    }
    fclose(file);

    return true;
}

int main() {
  // Matrix size of 1024 x 1024;
  int N = 1 << 10;
  clock_t startTime = clock();

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  clock_t endTime = clock();
  
  double timeGpu = ((double) (endTime - startTime)) / CLOCKS_PER_SEC; 

  printf("GPU DONE in %f secs\n",timeGpu);
  
  

  // Check result
  check_result(h_c, N);
  
  endTime = clock();
  double timeCpu = ((double) (endTime - startTime)) / CLOCKS_PER_SEC; 

  printf("CPU DONE in %f secs\n",timeCpu);

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}